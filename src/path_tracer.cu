#include "hip/hip_runtime.h"
#include "path_tracer.hpp"

#include <>

#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "constants.hpp"
#include "types.hpp"
#include "utils.hpp"

namespace crt {

std::ostream& operator<<(std::ostream& os, float3 value) {
  os << "( " << value.x << ", " << value.y << ", " << value.z << " )";
  return os;
}

namespace {

__device__ inline float3 ComputeReflectionDirection(const float3 normal,
                                                    const float3 incident) {
  /* Compute reflection direction.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 11" */
  return normalize(incident - 2.0 * dot(incident, normal) * normal);
}

__device__ inline float3 ComputeTransmissionDirection(const float3 normal,
                                                      const float3 incident,
                                                      float air_ior,
                                                      float material_ior) {
  /* Compute refraction direction according to Snell's Law.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 15" */

  float cos_theta_i = dot(normal, incident);

  float eta = 0;
  if (cos_theta_i >= 0) { /* Material -> Air */
    eta = material_ior / air_ior;
  } else { /* Air -> Material */
    eta = air_ior / material_ior;
  }

  float radicand = 1 - (eta * eta) * (1 - (cos_theta_i * cos_theta_i));
  if (radicand < 0) return make_float3(0.0f);  // No Refrection!!!
  float cos_theta_o = sqrt(radicand);

  if (cos_theta_i >= 0) {
    return normalize((eta * -cos_theta_i + cos_theta_o) * normal +
                     eta * incident);
  } else {
    return normalize((eta * -cos_theta_i - cos_theta_o) * normal +
                     eta * incident);
  }
}

__device__ float3 ComputeRandomCosineWeightedDirection(
    const float3 normal, hiprandState* curand_state) {
  /* Compute a random cosine weighted direction in heimsphere */
  float random1;
  random1 = hiprand_uniform(curand_state);
  float theta = kTwoPi * random1;

  float random2 = hiprand_uniform(curand_state);
  float cos_phi = sqrt(random2);
  float sin_phi = sqrt(1 - random2);

  /* Choose a axis not near to normal */
  float3 not_normal;
  if (fabs(normal.x) < kSQRTOfOneThird) {
    not_normal = make_float3(1, 0, 0);
  } else if (fabs(normal.y) < kSQRTOfOneThird) {
    not_normal = make_float3(0, 1, 0);
  } else {
    not_normal = make_float3(0, 0, 1);
  }

  float3 x_axis = cross(normal, not_normal);
  float3 y_axis = cross(normal, x_axis);

  return normalize((cos(theta) * sin_phi * x_axis) +
                   (sin(theta) * sin_phi * y_axis) + (cos_phi * normal));
}

__device__ ReflectionType RussianRoulette(const Material& material,
                                          hiprandState* state) {
  // TODO Always Diffuse
  float3 threshold[3];
  threshold[0] = material.diffuse_color;
  threshold[1] = threshold[0] + material.specular_color;
  threshold[2] = threshold[1] + (1 - material.dissolve);

  float3 random =
      threshold[2] * make_float3(hiprand_uniform(state), hiprand_uniform(state),
                                 hiprand_uniform(state));
  if (random <= threshold[0]) {
    return ReflectionType::DIFFUSE;
  } else if (random <= threshold[1]) {
    return ReflectionType::SPECULAR;
  } else {
    return ReflectionType::TRANSMISSION;
  }
}

__global__ void InitializationKernal(hiprandState* states, size_t num_pixels,
                                     size_t seed) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  hiprand_init(hash(idx) * hash(seed), 0, 0, &states[idx]);
}

__global__ void RayCastFromCameraKernel(const Camera camera, Ray* rays,
                                        size_t num_pixels,
                                        hiprandState* states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;
  size_t x = idx % camera.resolution.x;
  size_t y = camera.resolution.y - idx / camera.resolution.x - 1;

  hiprandState* const curand_state = &states[idx];

  // compute axis direction
  float3 x_axis = normalize(cross(camera.up, camera.view));
  float3 y_axis = normalize(cross(camera.view, x_axis));

  // compute size and center position of image plane
  // according to focal distance and fov
  float3 center = camera.position + camera.focal_distance * camera.view;
  float2 size =
      2 * make_float2(
              camera.focal_distance * tan((camera.fov.x / 2) * kArcPerAngle),
              camera.focal_distance * tan((camera.fov.y / 2) * kArcPerAngle));

  // compute the jittered point position on image plane
  // 1. generate random jitter offsets(in pixel) for supersample
  float2 jitter = make_float2(hiprand_uniform(curand_state) - 0.5,
                              hiprand_uniform(curand_state) - 0.5);
  // 2. compute distances to the center of image plane
  float2 distances = make_float2(make_uint2(x, y)) + jitter;
  distances /= (make_float2(camera.resolution) - 1);
  distances -= 0.5;
  distances *= size;
  // 3. compute point coordinate
  float3 point = center + x_axis * distances.x + y_axis * distances.y;

  // compute origin of the ray
  float3 origin = camera.position;
  if (camera.aperture_radius > kEpsilon) {
    // generate a random point on the aperture
    float angle = kTwoPi * hiprand_uniform(curand_state);
    float distance =
        camera.aperture_radius * sqrt(hiprand_uniform(curand_state));

    float2 coord = make_float2(cos(angle) * distance, sin(angle) * distance);

    origin += x_axis * coord.x + y_axis * coord.y;
  }

  // printf("\t%f, %f\n", (point).x, (point).y);
  rays[idx].origin = origin;
  rays[idx].direction = normalize(point - origin);
  rays[idx].color = make_float3(1);
}

// __global__ void RayTraceKernel(Scene scene, Ray* rays, int num_pixels);
__global__ void PathTraceKernel(const Triangle* triangles,
                                const size_t num_triangles, size_t* indices,
                                Ray* rays, float3* colors,
                                const size_t num_pixels,
                                hiprandState* curand_states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  size_t& index = indices[idx];
  if (index == kMaximumSize) return;

  Ray& ray = rays[index];
  hiprandState* curand_state = &curand_states[index];

  /** Get the nearest intersection */
  size_t intersection_idx = kInvalidIndex;
  float distance = 1e10f;

  for (size_t i = 0; i < num_triangles; i++) {
    float t = triangles[i].Hit(ray);
    if (t > 0 && t < distance) {
      distance = t;
      intersection_idx = i;
    }
  }

  /* TODO Air Absorption & Scattering */

  /** If no intersection, mark as dead ray */
  if (intersection_idx == kInvalidIndex) {
    index = kInvalidIndex;
  } else {
    /** Get secondary ray */
    const Triangle triangle = triangles[intersection_idx];

    if (triangle.material.Emit()) {
      float3 intersection = ray.origin + ray.direction * distance;
      colors[index] += (ray.color * triangle.material.emitted_color);
      index = kInvalidIndex;
    } else {
      ray.origin += ray.direction * distance;
      ray.origin += triangle.normal * 0.01f;

      ReflectionType type = RussianRoulette(triangle.material, curand_state);

      if (type == ReflectionType::TRANSMISSION) { /* refraction */
        ray.color *= triangle.material.dissolve;
        ray.direction = ComputeTransmissionDirection(
            triangle.normal, ray.direction, kAirIoR, triangle.material.ior);
      } else if (type == ReflectionType::SPECULAR) {
        ray.color *= triangle.material.specular_color;
        ray.direction =
            ComputeReflectionDirection(triangle.normal, ray.direction);
      } else if (type == ReflectionType::DIFFUSE) {
        ray.color *= triangle.material.diffuse_color;
        ray.direction =
            ComputeRandomCosineWeightedDirection(triangle.normal, curand_state);
        // printf("%d: %f, %f, %f\n", index, ray.direction.x, ray.direction.y,
        //       ray.direction.z);
      }
    }

    /** Remove the ray if its weight is very small */
    if (dot(ray.color, ray.color) <= 1e-2) {
      index = kInvalidIndex;
    }
  }
}

}  // namespace

Image PathTracer::Render(const Camera& camera) {
  const size_t num_pixels = camera.resolution.x * camera.resolution.y;

  thrust::device_vector<Triangle> triangles(m_scene.triangles);
  Triangle* const triangles_ptr = thrust::raw_pointer_cast(triangles.data());
  size_t const triangles_size = triangles.size();

  thrust::device_vector<float3> colors(num_pixels, make_float3(0));
  float3* const colors_ptr = thrust::raw_pointer_cast(colors.data());

  thrust::device_vector<Ray> rays(num_pixels);
  Ray* const rays_ptr = thrust::raw_pointer_cast(rays.data());

  thrust::device_vector<hiprandState> curand_states(num_pixels);
  hiprandState* const curand_states_ptr =
      thrust::raw_pointer_cast(curand_states.data());

  for (size_t counter = 0; counter < m_parameter.mc_sample_times; counter++) {
    /* Initialize hiprand */
    InitializationKernal<<<divUp(num_pixels, kThreadsPerBlock),
                           kThreadsPerBlock>>>(curand_states_ptr, num_pixels,
                                               counter);

    /* Create rays from camera */
    RayCastFromCameraKernel<<<divUp(num_pixels, kThreadsPerBlock),
                              kThreadsPerBlock>>>(camera, rays_ptr, num_pixels,
                                                  curand_states_ptr);

    /* Init indices */
    thrust::device_vector<size_t> indices(num_pixels);
    thrust::sequence(indices.begin(), indices.end());
    size_t* indices_ptr = thrust::raw_pointer_cast(indices.data());
    size_t indices_size = num_pixels;

    for (size_t depth = 0; depth < m_parameter.max_trace_depth; depth++) {
      // Step 1. trace rays to get secondary rays
      PathTraceKernel<<<divUp(indices.size(), kThreadsPerBlock),
                        kThreadsPerBlock>>>(triangles_ptr, triangles_size,
                                            indices_ptr, rays_ptr, colors_ptr,
                                            indices_size, curand_states_ptr);

      // Step 2. compact rays, remove dead rays
      thrust::device_vector<size_t>::iterator end = thrust::remove_if(
          indices.begin(), indices.end(), IsUnsignedMinusOne<size_t>());
      indices_size = end - indices.begin();
    }
  }

  return Image(camera.resolution, thrust::host_vector<float3>(colors),
               Color2Pixel(1));
  //              Color2Pixel(m_parameter.mc_sample_times));
}

}  // namespace crt