#include "hip/hip_runtime.h"
#include "path_tracer.hpp"

#include <>

#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "constants.hpp"
#include "types.hpp"
#include "utils.hpp"

namespace crt {

namespace {

__device__ inline float3 ComputeReflectionDirection(const float3 normal,
                                                    const float3 incident) {
  /* Compute reflection direction.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 11" */
  return normalize(incident - 2.0 * dot(incident, normal) * normal);
}

__device__ inline float3 ComputeTransmissionDirection(const float3 normal,
                                                      const float3 incident,
                                                      float air_ior,
                                                      float material_ior) {
  /* Compute refraction direction according to Snell's Law.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 15" */

  float cos_theta_i = dot(normal, incident);

  float eta = 0;
  if (cos_theta_i >= 0) { /* Material -> Air */
    eta = material_ior / air_ior;
  } else { /* Air -> Material */
    eta = air_ior / material_ior;
  }

  float radicand = 1 - (eta * eta) * (1 - (cos_theta_i * cos_theta_i));
  if (radicand < 0) return make_float3(0.0f);  // No Refrection!!!
  float cos_theta_o = sqrt(radicand);

  return normalize((eta * -cos_theta_i + sign(cos_theta_i) * cos_theta_o) *
                       normal +
                   eta * incident);
}

__device__ float3 ComputeRandomCosineWeightedDirection(
    const float3 normal, const float3 incident, hiprandState* curand_state) {
  /* Compute a random cosine weighted direction in heimsphere */
  float random1;
  random1 = hiprand_uniform(curand_state);
  float theta = kTwoPi * random1;

  float random2 = hiprand_uniform(curand_state);
  float cos_phi = sqrt(random2);
  float sin_phi = sqrt(1 - random2);

  /* Choose a axis not near to normal */
  float3 not_normal;
  if (fabs(normal.x) < kSQRTOfOneThird) {
    not_normal = make_float3(1, 0, 0);
  } else if (fabs(normal.y) < kSQRTOfOneThird) {
    not_normal = make_float3(0, 1, 0);
  } else {
    not_normal = make_float3(0, 0, 1);
  }

  float3 x_axis = cross(normal, not_normal);
  float3 y_axis = cross(normal, x_axis);

  return -sign(dot(normal, incident)) *
         normalize((cos(theta) * sin_phi * x_axis) +
                   (sin(theta) * sin_phi * y_axis) + (cos_phi * normal));
}

__device__ ReflectionType RussianRoulette(const float3 normal,
                                          const float3 incident,
                                          const Material& material,
                                          hiprandState* state) {
  float3 threshold[3];
  threshold[0] = material.diffuse_color;
  threshold[1] = threshold[0] +
                 material.specular_color * (dot(normal, incident) < 0 ? 1 : 0);
  threshold[2] = threshold[1] + (1 - material.dissolve);

  float3 random =
      threshold[2] * make_float3(hiprand_uniform(state), hiprand_uniform(state),
                                 hiprand_uniform(state));
  if (random <= threshold[0]) {
    return ReflectionType::DIFFUSE;
  } else if (random <= threshold[1]) {
    return ReflectionType::SPECULAR;
  } else {
    return ReflectionType::TRANSMISSION;
  }
}

__global__ void InitializationKernal(size_t* indices, hiprandState* states,
                                     size_t num_pixels, size_t seed) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  indices[idx] = idx;
  hiprand_init(hash(idx) * hash(seed), 0, 0, &states[idx]);
}

__global__ void RayCastFromCameraKernel(const Camera camera, Ray* rays,
                                        size_t num_pixels, unsigned light,
                                        hiprandState* states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;
  size_t x = idx % camera.resolution.x;
  size_t y = camera.resolution.y - idx / camera.resolution.x - 1;

  hiprandState* const curand_state = &states[idx];

  // compute axis direction
  float3 x_axis = normalize(cross(camera.view, camera.up));
  float3 y_axis = normalize(cross(x_axis, camera.view));

  // compute size and center position of image plane
  // according to focal distance and fov
  float3 center = camera.position + camera.focal_distance * camera.view;
  float2 size =
      2 * make_float2(
              camera.focal_distance * tan((camera.fov.x / 2) * kArcPerAngle),
              camera.focal_distance * tan((camera.fov.y / 2) * kArcPerAngle));

  // compute the jittered point position on image plane
  // 1. generate random jitter offsets(in pixel) for supersample
  float2 jitter = make_float2(hiprand_uniform(curand_state) - 0.5,
                              hiprand_uniform(curand_state) - 0.5);
  // 2. compute distances to the center of image plane
  float2 distances = make_float2(make_uint2(x, y)) + jitter;
  distances /= (make_float2(camera.resolution) - 1);
  distances -= 0.5;
  distances *= size;
  // 3. compute point coordinate
  float3 point = center + x_axis * distances.x + y_axis * distances.y;

  // compute origin of the ray
  float3 origin = camera.position;
  if (camera.aperture_radius > kEpsilon) {
    // generate a random point on the aperture
    float angle = kTwoPi * hiprand_uniform(curand_state);
    float distance =
        camera.aperture_radius * sqrt(hiprand_uniform(curand_state));

    float2 coord = make_float2(cos(angle) * distance, sin(angle) * distance);

    origin += x_axis * coord.x + y_axis * coord.y;
  }

  rays[idx].origin = origin;
  rays[idx].direction = normalize(point - origin);
  rays[idx].color = make_float3(light);
}

// __global__ void RayTraceKernel(Scene scene, Ray* rays, int num_pixels);
__global__ void PathTraceKernel(const Triangle* triangles,
                                const size_t num_triangles, size_t* indices,
                                Ray* rays, float3* colors,
                                const size_t num_pixels,
                                hiprandState* curand_states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  size_t& index = indices[idx];
  if (index == kMaximumSize) return;

  Ray& ray = rays[index];
  hiprandState* curand_state = &curand_states[index];

  /** Get the nearest intersection */
  size_t intersection_idx = kInvalidIndex;
  float3 weight = make_float3(1e10f); /* p, q, t */

  for (size_t i = 0; i < num_triangles; i++) {
    float3 w = triangles[i].Hit(ray);
    if (w.z > 0 && w.z < weight.z) {
      weight = w;
      intersection_idx = i;
    }
  }

  /** If no intersection, mark as dead ray */
  if (intersection_idx == kInvalidIndex) {
    index = kInvalidIndex;
  } else {
    /** Get secondary ray */
    const Triangle triangle = triangles[intersection_idx];
    const float distance = weight.z;

    ray.origin += distance * ray.direction;
    const float3 normal = triangle.GetNormal(weight.x, weight.y);

    ReflectionType type =
        RussianRoulette(normal, ray.direction, triangle.material, curand_state);
    if (type == ReflectionType::TRANSMISSION) {
      ray.direction = ComputeTransmissionDirection(
          normal, ray.direction, kAirIoR, triangle.material.ior);
    } else if (type == ReflectionType::SPECULAR) {
      ray.color *= triangle.material.specular_color;
      ray.direction = ComputeReflectionDirection(normal, ray.direction);
    } else if (type == ReflectionType::DIFFUSE) {
      colors[index] += (ray.color * triangle.material.emitted_color);

      ray.color *= triangle.material.diffuse_color;
      ray.direction = ComputeRandomCosineWeightedDirection(
          normal, ray.direction, curand_state);
    }
    ray.origin += kRayOriginBias * ray.direction;
  }
}

}  // namespace

Image PathTracer::Render(const Camera& camera) {
  const size_t num_pixels = camera.resolution.x * camera.resolution.y;

  thrust::device_vector<Triangle> triangles(m_scene.triangles);
  Triangle* const triangles_ptr = thrust::raw_pointer_cast(triangles.data());

  thrust::device_vector<float3> colors(num_pixels, make_float3(0));
  float3* const colors_ptr = thrust::raw_pointer_cast(colors.data());

  thrust::device_vector<Ray> rays(num_pixels);
  Ray* const rays_ptr = thrust::raw_pointer_cast(rays.data());

  thrust::device_vector<hiprandState> curand_states(num_pixels);
  hiprandState* const curand_states_ptr =
      thrust::raw_pointer_cast(curand_states.data());

  thrust::device_vector<size_t> indices(num_pixels);
  size_t* indices_ptr = thrust::raw_pointer_cast(indices.data());

  for (size_t counter = 0; counter < m_parameter.mc_sample_times; counter++) {
    indices.resize(num_pixels);

    /* Initialization */
    InitializationKernal<<<divUp(num_pixels, kThreadsPerBlock),
                           kThreadsPerBlock>>>(indices_ptr, curand_states_ptr,
                                               num_pixels, counter);

    /* Create rays from camera */
    RayCastFromCameraKernel<<<divUp(num_pixels, kThreadsPerBlock),
                              kThreadsPerBlock>>>(
        camera, rays_ptr, num_pixels, m_scene.light, curand_states_ptr);

    for (size_t depth = 0; depth < m_parameter.max_trace_depth; depth++) {
      // Step 0. Check if over.
      if (indices.size() == 0) break;

      // Step 1. Trace rays to get secondary rays.
      PathTraceKernel<<<divUp(indices.size(), kThreadsPerBlock),
                        kThreadsPerBlock>>>(triangles_ptr, triangles.size(),
                                            indices_ptr, rays_ptr, colors_ptr,
                                            indices.size(), curand_states_ptr);

      // Step 2. Compact rays, remove dead rays.
      thrust::device_vector<size_t>::iterator end = thrust::remove_if(
          indices.begin(), indices.end(), IsUnsignedMinusOne<size_t>());
      indices.resize(end - indices.begin());
    }
  }

  return Image(camera.resolution, thrust::host_vector<float3>(colors),
               // Color2Pixel(10));
               Color2Pixel(m_parameter.mc_sample_times));
}

}  // namespace crt