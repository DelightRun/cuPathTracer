#include "hip/hip_runtime.h"
#include "path_tracer.hpp"

#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>

#include <hiprand/hiprand_kernel.h>

#include "constants.hpp"
#include "types.hpp"
#include "utils.hpp"

namespace cupt {

namespace {

__constant__ Camera camera;

__device__ inline float3 ComputeReflectionDirection(const float3 normal,
                                                    const float3 incident) {
  /* Compute reflection direction.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 11" */
  return normalize(incident - 2.0 * dot(incident, normal) * normal);
}

__device__ float3 ComputeRandomCosineWeightedDirection(
    const float3 normal, const float3 incident, const float shininess,
    hiprandState* curand_state) {
  /* Compute a random cosine weighted direction in heimsphere */
  float random1 = hiprand_uniform(curand_state);
  float cos_phi = powf(random1, 1.0 / (1 + shininess));
  float sin_phi = sqrt(1 - square(cos_phi));

  float random2 = hiprand_uniform(curand_state);
  float theta = kTwoPi * random2;

  /* Choose a axis not near to normal */
  float3 not_normal;
  if (fabs(normal.x) < kSQRTOfOneThird) {
    not_normal = make_float3(1, 0, 0);
  } else if (fabs(normal.y) < kSQRTOfOneThird) {
    not_normal = make_float3(0, 1, 0);
  } else {
    not_normal = make_float3(0, 0, 1);
  }

  float3 x_axis = cross(normal, not_normal);
  float3 y_axis = cross(normal, x_axis);

  float3 direction = ((cos(theta) * sin_phi * x_axis) +
                      (sin(theta) * sin_phi * y_axis) + (cos_phi * normal));
  return normalize(direction);
}

__device__ inline float3 ComputeTransmissionDirection(
    const float3 normal, const float3 incident, const float eta,
    hiprandState* /* ignored */) {
  /* Compute refraction direction according to Snell's Law.
   * Refer to "Rui Wang, Lec12 - Ray Tracing, page 15" */
  float cos_theta_i = dot(normal, incident);

  float radicand = 1 - square(eta) * (1 - square(cos_theta_i));
  if (radicand < 0) /* Total Internal Reflection */
    return make_float3(0);
  float cos_theta_o = sqrt(radicand);

  float3 direction =
      (sign(cos_theta_i) * cos_theta_o - eta * cos_theta_i) * normal +
      eta * incident;
  return normalize(direction);
}

__device__ float3 ComputeSpecularityDirection(const float3 normal,
                                              const float3 incident,
                                              const float shininess,
                                              hiprandState* curand_state) {
  float3 perfect = ComputeReflectionDirection(normal, incident);
  return ComputeRandomCosineWeightedDirection(perfect, incident, shininess,
                                              curand_state);
}

__device__ float3 ComputeDiffusionDirection(const float3 normal,
                                            const float3 incident,
                                            hiprandState* curand_state) {
  return ComputeRandomCosineWeightedDirection(normal, incident, 1.0,
                                              curand_state);
}

__global__ void InitializationKernal(size_t* indices,
                                     hiprandState* curand_states,
                                     size_t num_pixels, size_t seed) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  indices[idx] = idx;
  hiprand_init(hash(idx) * hash(seed), 0, 0, &curand_states[idx]);
}

__global__ void RayCastFromCameraKernel(Ray* rays, const size_t num_pixels,
                                        const float intensity,
                                        hiprandState* curand_states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  size_t x = idx % camera.resolution.x;
  size_t y = camera.resolution.y - idx / camera.resolution.x - 1;

  hiprandState* const curand_state = &curand_states[idx];

  /* compute axis direction */
  float3 x_axis = normalize(cross(camera.view, camera.up));
  float3 y_axis = normalize(cross(x_axis, camera.view));

  /* compute image plane ratio and center position */
  float ratio = camera.resolution.x * 1.0 / camera.resolution.y;
  float3 center = camera.position + camera.view * camera.focal_distance;

  /* compute the jittered point position on image plane */
  float2 jitter = make_float2(hiprand_uniform(curand_state) - 0.5,
                              hiprand_uniform(curand_state) - 0.5);
  float2 distances = (make_float2(make_uint2(x, y)) + jitter) /
                     (make_float2(camera.resolution) - 1);
  distances = (2 * distances - 1) * make_float2(ratio, 1);
  float3 point = center + distances.x * x_axis + distances.y * y_axis;

  /* compute origin of the ray */
  float3 origin = camera.position;
  if (camera.aperture_radius > kEpsilon) {
    float angle = kTwoPi * hiprand_uniform(curand_state);
    float distance =
        camera.aperture_radius * sqrt(hiprand_uniform(curand_state));
    float2 coord = make_float2(cos(angle) * distance, sin(angle) * distance);
    origin += x_axis * coord.x + y_axis * coord.y;
  }

  rays[idx].origin = origin;
  rays[idx].direction = normalize(point - origin);
  rays[idx].color = make_float3(intensity);
}

__global__ void PathTraceKernel(const Triangle* triangles,
                                const size_t num_triangles, size_t* indices,
                                Ray* rays, float3* colors,
                                const size_t num_pixels,
                                hiprandState* curand_states) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pixels) return;

  size_t& index = indices[idx];
  if (index == kMaximumSize) return;

  Ray& ray = rays[index];
  hiprandState* curand_state = &curand_states[index];

  /** Get the nearest intersection */
  size_t intersection_idx = kInvalidIndex;
  float3 weight = make_float3(1e10f); /* p, q, t */

  for (size_t i = 0; i < num_triangles; i++) {
    float3 w = triangles[i].Hit(ray);
    if (w.z > 0 && w.z < weight.z) {
      weight = w;
      intersection_idx = i;
    }
  }

  /** If no intersection, mark as dead ray */
  if (intersection_idx == kInvalidIndex) {
    index = kInvalidIndex;
  } else { /** Else get secondary ray */
    /* Transmit ray to the intersection point */
    ray.origin += weight.z * ray.direction;

    const Triangle triangle = triangles[intersection_idx];
    float3 normal = triangle.GetNormal(weight.x, weight.y);
    float shininess = triangle.material.shininess;
    float eta = 1.0;
    bool into = (dot(normal, ray.direction) < 0);
    if (!into) normal *= -1;

    /* Specular material by default */
    float3 diffusion = triangle.material.diffuse_color;
    float3 specularity = triangle.material.specular_color;
    float3 transmission = make_float3(0);

    /* Transparent material, calculate fresnel cofficient */
    if (triangle.material.dissolve < 1) {
      float incident_ior = kAirIoR, transmitted_ior = kAirIoR;
      if (into) /* Air -> Material */
        transmitted_ior = triangle.material.ior;
      else /* Material -> ir */
        incident_ior = triangle.material.ior;
      eta = incident_ior / transmitted_ior;
      shininess = 1000;  // Mirror reflection

      const float3 direction = ComputeTransmissionDirection(
          normal, ray.direction, eta, curand_state);

      if (iszero(direction)) { /* Total Internal Reflection */
        specularity = make_float3(1);
        transmission = make_float3(0);
      } else { /* Calculate Fresnel Cofficient */
        float cos_theta_i = fabs(dot(normal, ray.direction));
        float cos_theta_o = fabs(dot(normal, direction));
        float rs = square(
            (incident_ior * cos_theta_i - transmitted_ior * cos_theta_o) /
            (incident_ior * cos_theta_i + transmitted_ior * cos_theta_o));
        float rt = square(
            (incident_ior * cos_theta_o - transmitted_ior * cos_theta_i) /
            (incident_ior * cos_theta_o + transmitted_ior * cos_theta_i));
        float r = (rs + rt) / 2;

        specularity = make_float3(r);
        transmission = make_float3(1 - r);
      }
    }

    /* Russian Roulette */
    float3 threshold[3];
    threshold[0] = diffusion;
    threshold[1] = threshold[0] + specularity;
    threshold[2] = threshold[1] + transmission;
    float3 random = threshold[2] * make_float3(hiprand_uniform(curand_state),
                                               hiprand_uniform(curand_state),
                                               hiprand_uniform(curand_state));

    if (random <= threshold[0]) { /* Diffusion */
      colors[index] += (ray.color * triangle.material.emitted_color);
      ray.color *= triangle.material.diffuse_color;
      ray.direction =
          ComputeDiffusionDirection(normal, ray.direction, curand_state);
    } else if (random <= threshold[1]) { /* Specular */
      ray.color *= triangle.material.specular_color;
      ray.direction = ComputeSpecularityDirection(normal, ray.direction,
                                                  shininess, curand_state);
    } else if (random <= threshold[2]) { /* Transmission */
      ray.color *= (1 - triangle.material.dissolve);
      ray.direction = ComputeTransmissionDirection(normal, ray.direction, eta,
                                                   curand_state);
    }
    ray.origin += kRayOriginBias * ray.direction;
  }
}

}  // namespace

Image PathTracer::Render(const Camera& host_camera) {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(camera), &host_camera, sizeof(Camera)));

  const size_t num_pixels = host_camera.resolution.x * host_camera.resolution.y;

  thrust::device_vector<Triangle> triangles(m_scene.triangles);
  Triangle* const triangles_ptr = thrust::raw_pointer_cast(triangles.data());

  thrust::device_vector<float3> colors(num_pixels, make_float3(0));
  float3* const colors_ptr = thrust::raw_pointer_cast(colors.data());

  thrust::device_vector<Ray> rays(num_pixels);
  Ray* const rays_ptr = thrust::raw_pointer_cast(rays.data());

  thrust::device_vector<hiprandState> curand_curand_states(num_pixels);
  hiprandState* const curand_curand_states_ptr =
      thrust::raw_pointer_cast(curand_curand_states.data());

  thrust::device_vector<size_t> indices(num_pixels);
  size_t* indices_ptr = thrust::raw_pointer_cast(indices.data());

  for (size_t counter = 0; counter < m_parameter.mc_sample_times; counter++) {
    /* Initialization */
    indices.resize(num_pixels);
    InitializationKernal<<<divUp(num_pixels, kThreadsPerBlock),
                           kThreadsPerBlock>>>(
        indices_ptr, curand_curand_states_ptr, num_pixels, counter);

    /* Create rays from camera */
    RayCastFromCameraKernel<<<divUp(num_pixels, kThreadsPerBlock),
                              kThreadsPerBlock>>>(
        rays_ptr, num_pixels, m_scene.intensity, curand_curand_states_ptr);

    for (size_t depth = 0; depth < m_parameter.max_trace_depth; depth++) {
      /* Step 0. Check if over. */
      if (indices.size() == 0) break;

      /* Step 1. Trace rays to get secondary rays. */
      PathTraceKernel<<<divUp(indices.size(), kThreadsPerBlock),
                        kThreadsPerBlock>>>(
          triangles_ptr, triangles.size(), indices_ptr, rays_ptr, colors_ptr,
          indices.size(), curand_curand_states_ptr);

      /* Step 2. Compact rays, remove dead rays. */
      thrust::device_vector<size_t>::iterator end =
          thrust::remove_if(indices.begin(), indices.end(), IsInvalidIndex());
      indices.resize(end - indices.begin());
    }
  }

  return Image(host_camera.resolution, colors,
               Color2Pixel(m_parameter.mc_sample_times));
}

}  // namespace cupt